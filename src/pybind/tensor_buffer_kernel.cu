#include <hip/hip_runtime.h>

#include <stdio.h>
#include <vector>

template <typename scalar_t>
__global__ void cuda_increment_kernel(scalar_t *__restrict__ array, int *__restrict__ offsets, const size_t size) {
    int index = threadIdx.x; // Pixel offset
    if (index < size) {
      float val = 1;
      atomicAdd((array + offsets[index]), val);
    }
}

void index_increment_cuda(float *array, int *offset_pointer, size_t indices, int* event_device_pointer) {
  const size_t buffer_size = indices * sizeof(int);

  hipMemcpyAsync(event_device_pointer, offset_pointer, buffer_size, hipMemcpyHostToDevice, 0);
  cuda_increment_kernel<float><<<1, indices>>>(array, event_device_pointer, indices);
}

void* alloc_memory_cuda(size_t buffer_size, size_t bytes) {
  void *cuda_device_pointer;
  const size_t size = buffer_size * bytes;
  hipMallocAsync(&cuda_device_pointer, size, 0);
  hipMemsetAsync(&cuda_device_pointer, 0, size, 0);
  return cuda_device_pointer;
}

void free_memory_cuda(void* cuda_device_pointer) {
  hipFreeAsync(cuda_device_pointer, 0);
}